

#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;


__global__ void matrix_mul(int *A, int *B, int *C, int N , int M , int P, int n, int core){
    // n -> number of matrix
    int i = threadIdx.x; // koy number thread

    int start_index = i * (n / core);
    int end_index = (i + 1) * (n / core);
    if(i == core - 1){
        end_index = n;
    }
    for(int x  = start_index; x < end_index; x++){
        int add_a = x * N * M;
        int add_b = x * M * P;
        int add_c = x * N * P;

        for(int i = 0; i < N;i++){
            for(int j =0 ; j < P ; j++){
                int sum = 0;
                for(int k = 0; k < M; k++){
                    //sum += A[i][k] * B[k][j];
                    // sum += A[i * M + k] * B [k * P + j];
                    sum += A[add_a + i * M + k] * B[add_b + k * P + j];
                }
                // C[i][j] = sum;
                C[add_c + i * P + j] = sum;
            }
        }
    }

}
int main(int argc, char* argv[]){
    int N , M , P;
        cout << "Enter the dimenssion of matrices...." << endl;
        cin >> N >> M >> P;

        int number_of_matrix; 
        cout << "Enter the number of matrix" << endl; 
        cin >> number_of_matrix;

        cout << "Enter matrix A: " << endl;

        int *A;
        int a_element = number_of_matrix * N * M;

        A = new int[a_element];
        for(int i = 0; i < a_element ; i++){
            //cin >> A[i];  // jodi input file thake.
             A[i] = 1;

        }
        cout << "Enter the matrix B: " << endl;

        int b_element = number_of_matrix * M * P;
        int *B;
        B = new int[b_element];
        for(int i = 0; i < b_element; i++){
            //cin >> B[i];
            B[i] = 1;
        }
        // cout << "Input taken" << endl;
        // input done;

        int *C;
        int c_element = number_of_matrix * N * P;

        C = new int[c_element];

        // A, B C ram array
        // DA, DB, DC device array
      int *DA, *DB, *DC;
        hipMalloc(&DA, a_element * sizeof(int));
        hipMalloc(&DB, b_element * sizeof(int));
        hipMalloc(&DC, c_element * sizeof(int));

        hipMemcpy(DA, A, a_element * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(DB, B, b_element * sizeof(int), hipMemcpyHostToDevice);

        int number_of_core = 2;
        cout << "enter number of core: " << endl;
        cin >> number_of_core;
        hipEvent_t start, end;
        hipEventCreate(&start);
        hipEventCreate(&end);
        hipEventRecord(start);
        matrix_mul<<<1, number_of_core>>> (DA, DB, DC, N , M, M, number_of_matrix, number_of_core);
// fuction name<<<block, ekta block e koto core thakbe>>>
        hipDeviceSynchronize();
        hipEventRecord(end);
        hipEventSynchronize(end);
        hipMemcpy(C, DC, c_element * sizeof(int), hipMemcpyDeviceToHost);

        float mili;
        cout << "function done" << endl;

        hipEventElapsedTime(&mili, start, end);

        cout << mili << " time passed" << endl;
        for(int i = 0;i < N; i++){
            for(int j = 0; j < P; j++){
                cout << C[i * P + j] << " ";
            }
            cout << endl;
        }


}