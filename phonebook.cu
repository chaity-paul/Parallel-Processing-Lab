
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;

__global__ void string_matching(char *text, char *pattern,int *index, int p_len, int n , int core){
    int i = threadIdx.x;
    int start_index = i * (n / core);
    int end_index = (i + 1) * (n / core);
    if(i == core -1){
        end_index = n;
    }

    for(int i = start_index; i < end_index; i++){
        int cnt = 0;
        for(int j = index[i]; text[j] != '\n'; j++){
            cnt = 0;
            for(int k = 0; k < p_len; k++){
                if(pattern[k] == text[k + j]){
                    cnt++;
                }else if((pattern[k] - 32) == text[k + j]){
                    cnt++;
                }
                else break;
            }
            if(cnt == p_len){
                index[i] = -1;
                break;
            }
        }
    }
}
int main(int argc, char *argv[]){
    vector<string> contact_list;
    string pattern = argv[1];
    int m = pattern.size();
    for(int i = 0; i < m ;i++){
        pattern[i] = tolower(pattern[i]);
    }

    for(int i = 2; i < argc; i++){
        ifstream buf(argv[i]); // taking the  file
        string contact;
        while(getline(buf, contact)){
            contact_list.push_back(contact); // take input  from the file
        }
    }
    
    int n = contact_list.size();
   
    vector<int> index(n);
    string text = "";

    for(int i = 0; i < n; i++){
        index[i] = text.size();
        text += contact_list[i] + "\n";
    }

    char *DT, *DP;
    hipMalloc(&DT, (text.size() + 1) * sizeof(char));
    hipMemcpy(DT, &text[0], (text.size() + 1) * sizeof(char), hipMemcpyHostToDevice);
    // copy Text to DT

    hipMalloc(&DP, (pattern.size() + 1) * sizeof(char));
    hipMemcpy(DP, &pattern[0], (pattern.size() + 1) * sizeof(char), hipMemcpyHostToDevice);
    // copy pattern to DP

    int *DI; // device index
    hipMalloc(&DI, n * sizeof(int));
    hipMemcpy(DI, &index[0], n * sizeof(int), hipMemcpyHostToDevice);
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    int number_of_core = 2;
    cout << "Enter the number of core: " << endl;
    cin >> number_of_core;

    string_matching<<<1 , number_of_core>>>(DT, DP, DI, pattern.size(), n, number_of_core);
    hipDeviceSynchronize();
    hipEventRecord(end);
    hipEventSynchronize(end);
    hipMemcpy(&index[0], DI, n * sizeof(int), hipMemcpyDeviceToHost);

    float mili;
    cout << "function done" << endl;

    hipEventElapsedTime(&mili, start, end);

    cout << mili << " time passed" << endl;
    set<string> ans;
    for(int i = 0; i < n; i++){
        if(index[i] == -1){
            ans.insert(contact_list[i]);
        }
    }
    for(set<string>::iterator it = ans.begin(); it != ans.end(); it++){
        cout << *it << endl;
    }
    
}